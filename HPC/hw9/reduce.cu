
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 160
#define THREADS 16


__global__ void reduce_kernel(float *A, float *cols)
{
  extern __shared__ float sdata[];
  sdata[threadIdx.x] = A[threadIdx.x];

  // do reduce in shared memory
  for(unsigned int s = blockDim.x/2; s>0; s>>=1)
  {
    if(threadIdx.x <s) sdata[threadIdx.x] += sdata[threadIdx.x+s];
    __syncthreads();
  }

  // write resul for this block to global memory
  if(threadIdx.x ==0) *cols = sdata[0];
}



int main(){


  float A[N], *cols, sum;
  float *cols_d, *A_d;
  int i;

  dim3 dimBlock(THREADS);
  dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x );



  for(i=0; i<N; i++){
    A[i] = i+1;  //1~160 claculation i+1
  }
  
  int sdata = THREADS * sizeof(float);
  cols = (float*)malloc(sizeof(float)*dimGrid.x);
  //cols = (float*)malloc(sizeof(float));

  hipMalloc( (void **) &cols_d, sizeof(float)*dimGrid.x );
  hipMalloc( (void **) &A_d, sizeof(float)*N );

  hipMemcpy(A_d, A, sizeof(float)*N, hipMemcpyHostToDevice);
  reduce_kernel<<<dimGrid, dimBlock, sdata>>>(A_d, cols_d);

  //copy values from Gpu memory to Cpu memory
  hipMemcpy(cols, cols_d, sizeof(float)*dimGrid.x, hipMemcpyDeviceToHost);

  //add the computed value to the value of pi
  // sum = *cols;

  sum = 0.0;
  for(i=0; i<dimGrid.x; i++)
   sum +=cols[i];
 
  printf("%f\n",sum);
  
  hipFree(A_d);
  hipFree(cols_d);


}



#include "hip/hip_runtime.h"
#include <stdio.h>

#define L 40
#define N 25600000
#define THREADS 512

__global__ void pi_kernel(float *cols, float p1, float p2)
{
  unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
  float xval, height, width;

  extern __shared__ float reduce[];

  // calculate each segement
  /* FILL */
    xval = 0.0;
    width = (p2-p1)/N;
    xval =p1+(width)*i;
    height = sqrt(1.0-xval*xval);
    reduce[threadIdx.x]= width*height;

  __syncthreads();

  // do reduce in shared memory
  /* FILL */
  for(unsigned int s = blockDim.x/2 ; s>0; s>>=1)
  {
    if(threadIdx.x<s)reduce[threadIdx.x] += reduce[threadIdx.x+s];
    __syncthreads();

  }
 
  // write resul for this block to global memory
  /* FILL */
  if(threadIdx.x==0)cols[blockIdx.x] =reduce[0];   

}

extern int millisecond();

int main()
{

  double pi;
  float *cols, *cols_d, p1, p2;
  int start, end;
  int i;
  int j;

  dim3 dimBlock(THREADS);
  dim3 dimGrid( (N+dimBlock.x-1)/dimBlock.x );
  
  int reduce = THREADS * sizeof(float);
  cols = (float*)malloc(sizeof(float)*dimGrid.x);

  start = millisecond();

  hipMalloc( (void **) &cols_d, sizeof(float)*dimGrid.x );

  p1 = 0.0;
  p2 = 1.0/(float)L;

  pi = 0.0;

  for( i=0; i<L; i++){
    pi_kernel<<<dimGrid, dimBlock, reduce>>>(cols_d, p1, p2);
  
    //copy values from GPU memory to CPU memory
    /* FILL */
    hipMemcpy(cols, cols_d, sizeof(float)*dimGrid.x, hipMemcpyDeviceToHost);
   
   //add the computed value to the value of pi
   /*  FILL */  
 
   for(j=0; j<dimGrid.x; j++){
	pi += cols[j];
   }

   p1 =p2;
   p2 = p1 + 1.0/(float)L;

  }
  
  hipDeviceSynchronize();

  end = millisecond();
  
  printf("pi = %17.15f\n", pi*4.0);
  printf("Elapsed time: %d milliseconds\n", end - start);

  hipFree(cols_d);

}




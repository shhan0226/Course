#define N 10000

#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS 10


__global__ void vecAdd(float * A, float *B, float *C)
{
  int i = blockDim.x*blockIdx.x+threadIdx.x;
  C[i] = A[i] + B[i];
}

int main()
{
  float A[N], B[N], C[N], C2[N];
  int i;
  float *A_d, *B_d, *C_d;

  dim3 dimBlock(THREADS);


  dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);

  for(i=0; i<N; i++){
    A[i] = i*2;
    B[i] = N-i;
    C2[i] = A[i] + B[i];
  }
  /*Allocate array on device*/
  hipMalloc((void**) &A_d, sizeof(float)*N);
  hipMalloc((void**) &B_d, sizeof(float)*N);
  hipMalloc((void**) &C_d, sizeof(float)*N);
  
  /*Copy array from host memory to device memory  */
  hipMemcpy(A_d, A, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, sizeof(float)*N, hipMemcpyHostToDevice);

  vecAdd<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);

  /*Copy the result back*/
  hipMemcpy(C, C_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  for(i=0; i<N; i++)
    if(C[i] != C2[i]) printf("%d %f %f\n", i, C[i], C2[i]);

  printf("OK!!!\n");
  /*Free meomory on the device*/
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}


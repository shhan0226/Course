
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 400
#define THREADS 16
#define A(i, j) A[i*N+j]
#define B(i, j) B[i*N+j]
#define C(i, j) C[i*N+j]

__global__ void matAdd(float *A, float *B, float *C)
{
  unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
  unsigned int j = blockIdx.y*blockDim.y+threadIdx.y;

  C(i,j) = A(i,j) + B(i,j);
}

int main()
{

  float A[N][N], B[N][N], C[N][N], C2[N][N];
  float *A_d, *B_d, *C_d;
  int i, j;
  
  dim3 dimBlock(THREADS, THREADS);
  dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,((N+dimBlock.y-1)/dimBlock.y));

  for(i=0; i<N; i++){
    for(j=0; j<N; j++){
      A[i][j] = i*j;
      B[i][j] = i+j;
      C2[i][j] = A[i][j] + B[i][j];
    }
  }  
  
  /*Allocate array on device*/
  hipMalloc((void**) &A_d, sizeof(float)*N*N);
  hipMalloc((void**) &B_d, sizeof(float)*N*N);
  hipMalloc((void**) &C_d, sizeof(float)*N*N);

  /*Copy array from host memory to device memory*/
  hipMemcpy(A_d, A, sizeof(float)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, sizeof(float)*N*N, hipMemcpyHostToDevice);

  matAdd<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);

  /*Copy the result back*/
  hipMemcpy(C, C_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);

  for( i=0; i<N; i++)
    for(j=0; j<N; j++)
      if(C[i][j] != C2[i][j]) printf("%d %f %f\n", i, C[i][j], C2[i][j]);

   printf("OK!!\n");

  /*Free memory on the device */
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

}
